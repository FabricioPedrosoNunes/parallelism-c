#include <stdio.h>
#include <hip/hip_runtime.h>

#define THREADSPERBLOCK 1024

__global__ void primeiroLaco(long int* d_num, long int* d_den, long int start, long int end, int size)
{
  int num_aux, den_aux, aux, resto;
  long int factor, ii, sum, done, n;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < size)
  {
    ii     = i - start;
    sum    = 1 + i;
    done   = i;
    factor = 2;

    while (factor < done)
    {
      resto = i / factor;
      resto = i - (factor * resto);

      if (resto == 0)
      {
        sum += (factor + (i / factor));

		    if ((done = i / factor) == factor)
		      sum -= factor;
      }

	    factor++;
    }

    d_num[ii] = sum;
    d_den[ii] = i;

    num_aux = d_num[ii];
    den_aux = d_den[ii];

  	while (num_aux != 0)
  	{
      aux     = num_aux; 
	    resto   = den_aux / num_aux;
  	  resto   = den_aux - (num_aux * resto);
	    num_aux = resto;  
	    den_aux = aux;
	  }

    n          = den_aux;
    d_num[ii] /= n;
    d_den[ii] /= n;
  }
}

__global__ void segundoLaco(long int* d_num, long int* d_den, long int* d_arrSomaC, int size)
{
  int j, i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < size)
  {
    for (j = i + 1; j < size; j++)
    {
      if ((d_num[i] == d_num[j]) && (d_den[i] == d_den[j]))
        d_arrSomaC[i]++;
    }
  }
}

void friendly_numbers(long int start, long int end)
{
  hipSetDevice(0);

  long int *d_num, *d_den, last = end - start + 1;
  size_t   size = last * sizeof(long int);
  int      c    = 0;

  int tamanho = end - start;
  int nBlocks = (tamanho + THREADSPERBLOCK - 1) / THREADSPERBLOCK;

  printf("BLOCOS %d THREADS %d\n", nBlocks, THREADSPERBLOCK);

  long int *num;
  num = (long int*) malloc(size);

  long int *den;
  den = (long int*) malloc(size);

  long int *arrSomaC;
  arrSomaC = (long int*) malloc(size);

  long int i;

  for (i = 0; i < last; i++)
    arrSomaC[i] = 0;

  hipMalloc((void**)&d_num, size);
  hipMalloc((void**)&d_den, size);

  hipMemcpy(d_num, num, size, hipMemcpyHostToDevice);
  hipMemcpy(d_den, den, size, hipMemcpyHostToDevice);

  primeiroLaco<<<nBlocks, THREADSPERBLOCK>>>(d_num, d_den, start, end, last);

  long int *d_arrSomaC;

  hipMalloc((void**)&d_arrSomaC, size);
  hipMemcpy(d_arrSomaC, arrSomaC, size, hipMemcpyHostToDevice);

  segundoLaco<<<nBlocks, THREADSPERBLOCK>>>(d_num, d_den, d_arrSomaC, last);

  hipMemcpy(arrSomaC, d_arrSomaC, size, hipMemcpyDeviceToHost);

  for (i = 0; i < last; i++)
    c += arrSomaC[i];

  printf("Founded %d pairs of mutually friendly numbers\n", c);

  hipFree(d_num);
  hipFree(d_den);
  hipFree(d_arrSomaC);

  free(num);
  free(den);
  free(arrSomaC);
}

int main(int argc, char **argv)
{
  long int start;
  long int end;

  start = atoi(argv[1]);
  end   = atoi(argv[2]);

  printf("NUMBER %ld TO %ld\n", start, end);
  friendly_numbers(start, end);

  return EXIT_SUCCESS;
}